//
// Created by SpaceEye on 12.06.22.
//

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "get_device_count.cuh"

int get_device_count() {
    int count;
    hipGetDeviceCount(&count);
    return count;
}